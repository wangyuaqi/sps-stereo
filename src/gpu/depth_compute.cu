#include "depth_compute.h"
#include "configuration.h"
#include "image_warp.h"
#include "costs.h"
#include "new_cost_aggregation.h"
#include <stdint.h>

using namespace std;
cv::Mat DepthComputeUtil::DepthCompute(pair<cv::Mat, cv::Mat> &right_right_pair,float disp_max, float disp_min) {

    float *right_right_disparity;
    const int depth_num = NEW_MAX_DISPARITY;
    hipMallocManaged((void**)&right_right_disparity,depth_num*sizeof(float));
    const float disp_dis = (disp_max-disp_min)/NEW_MAX_DISPARITY;
    for(int depth_iter = 0;depth_iter<depth_num;depth_iter++) {
        //cur_depth = depth_min + depth_iter * depth_dis;
        //left_left_disparity[depth_iter] = depth_min+disp_dis*depth_iter;
        right_right_disparity[depth_iter] = disp_min+disp_dis*depth_iter;
    }


    /*uint8_t *left_left_data;
    uint8_t *left_left_center_data;
    uint8_t *left_center_data;
    uint8_t *left_center_center_data;
    uint8_t *right_center_data;
    uint8_t *right_center_center_data;*/
    uint8_t *right_right_data;
    uint8_t *right_right_center_data;

    /*uint8_t **left_left_warp_data;
    uint8_t **left_center_warp_data;
    uint8_t **right_center_warp_data;*/
    uint8_t **right_right_warp_data;

    const int image_width = (right_right_pair.first).cols;
    const int image_height = (right_right_pair.first).rows;

    const int origin_image_size = image_width*image_height*sizeof(uint8_t);
    /*CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_left_data,origin_image_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_left_center_data,origin_image_size));

    CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_center_data,origin_image_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_center_center_data,origin_image_size));

    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_center_data,origin_image_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_center_center_data,origin_image_size));*/

    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_right_data,origin_image_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_right_center_data,origin_image_size));

    for(int image_x = 0;image_x<image_width;image_x++){
        for(int image_y = 0;image_y<image_height;image_y++){
            int image_index = image_y*image_width+image_x;
            /*left_left_data[image_index] = left_left_pair.second.at<uint8_t>(image_y,image_x);
            left_left_center_data[image_index] = left_left_pair.first.at<uint8_t>(image_y,image_x);

            left_center_data[image_index] = left_center_pair.second.at<uint8_t>(image_y,image_x);
            left_center_center_data[image_index] = left_center_pair.first.at<uint8_t>(image_y,image_x);

            right_center_data[image_index] = right_center_pair.second.at<uint8_t>(image_y,image_x);
            right_center_center_data[image_index] = right_center_pair.first.at<uint8_t>(image_y,image_x);*/

            right_right_data[image_index] = right_right_pair.second.at<uint8_t>(image_y,image_x);
            right_right_center_data[image_index] = right_right_pair.first.at<uint8_t>(image_y,image_x);
        }
    }
    cv::imwrite("4_base.png",right_right_pair.first);

    // allocate warped image memory
    std::cout<<"init"<<std::endl;
    const int dis_num = NEW_MAX_DISPARITY;

    const int warp_image_size = dis_num*sizeof(uint8_t*);
    /*CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_left_warp_data,warp_image_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_center_warp_data,warp_image_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_center_warp_data,warp_image_size));*/
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_right_warp_data,warp_image_size));


    for(int dis_iter =0;dis_iter<dis_num;dis_iter++){
        /*CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_left_warp_data[dis_iter],origin_image_size));
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_center_warp_data[dis_iter],origin_image_size));
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_center_warp_data[dis_iter],origin_image_size));*/
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_right_warp_data[dis_iter],origin_image_size));
    }
    hipStream_t n_stream1,n_stream2,n_stream3,n_stream4;
    CUDA_CHECK_RETURN(hipStreamCreate(&n_stream1));
    CUDA_CHECK_RETURN(hipStreamCreate(&n_stream2));
    CUDA_CHECK_RETURN(hipStreamCreate(&n_stream3));
    CUDA_CHECK_RETURN(hipStreamCreate(&n_stream4));

    const int w_size=NEW_MAX_DISPARITY;
    dim3 block_grid;
    block_grid.x=IMG_HEIGHT;
    block_grid.y=w_size;


    dim3 thread_grid;
    //thread_grid.x=IMG_WIDTH;
    thread_grid.x=IMG_WIDTH/2;

    hipStream_t w_stream1,w_stream2,w_stream3,w_stream4;
    CUDA_CHECK_RETURN(hipStreamCreate(&w_stream1));
    CUDA_CHECK_RETURN(hipStreamCreate(&w_stream2));
    CUDA_CHECK_RETURN(hipStreamCreate(&w_stream3));
    CUDA_CHECK_RETURN(hipStreamCreate(&w_stream4));

    hipError_t err;
    /*Image_Shift<<<block_grid,thread_grid,0>>>(left_center_data,left_center_warp_data,left_center_disparity,-1,0);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }

    Image_Shift<<<block_grid,thread_grid,0>>>(left_left_data,left_left_warp_data,left_left_disparity,-1,0);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }


    Image_Shift<<<block_grid,thread_grid,0>>>(right_center_data,right_center_warp_data,right_center_disparity,1,0);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }*/

    Image_Shift<<<block_grid,thread_grid,0>>>(right_right_data,right_right_warp_data,right_right_disparity,1,0);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }

    /*CUDA_CHECK_RETURN(hipStreamSynchronize(w_stream1));
    CUDA_CHECK_RETURN(hipStreamSynchronize(w_stream2));
    CUDA_CHECK_RETURN(hipStreamSynchronize(w_stream3));
    CUDA_CHECK_RETURN(hipStreamSynchronize(w_stream4));*/
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    //***********warp image output
    uint8_t *image_data;
    image_data = new uint8_t[image_height*image_width];
    for(int img_y = 0;img_y<image_height;img_y++){
        for(int img_x =0;img_x<image_width;img_x++){
            int img_index = img_x+img_y*image_width;
            image_data[img_index] = right_right_warp_data[115][img_index];
            //image_data[img_index] = left_left_center_data[img_index];
        }
    }
    cv::Mat warp_image(image_height,image_width,CV_8UC1,image_data);
    cv::imwrite("warp.png",warp_image);

    /*uint8_t *image_data_2;
    image_data_2 = new uint8_t[image_height*image_width];
    for(int img_y = 0;img_y<image_height;img_y++){
        for(int img_x =0;img_x<image_width;img_x++){
            int img_index = img_x+img_y*image_width;
            //image_data[img_index] = left_left_warp_data[120][img_index];
            image_data_2[img_index] = left_left_center_data[img_index];
        }
    }
    cv::Mat warp_image_2(image_height,image_width,CV_8UC1,image_data_2);
    cv::imwrite("warp_2.png",warp_image_2);*/

    //****************************

    // compute census transform
    /*cost_t **left_left_warp_census;
    cost_t *left_left_center_census;

    cost_t **left_center_warp_census;
    cost_t *left_center_center_census;

    cost_t **right_center_warp_census;
    cost_t *right_center_center_census;*/

    cost_t **right_right_warp_census;
    cost_t *right_right_center_census;



    //allocate warp census
    const int image_census_size = image_height*image_width*sizeof(cost_t);
    const int warp_census_size = dis_num*sizeof(cost_t*);

    /*CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_left_warp_census,warp_census_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_center_warp_census,warp_census_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_center_warp_census,warp_census_size));*/
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_right_warp_census,warp_census_size));


    for(int dis_iter =0;dis_iter<dis_num;dis_iter++)
    {
        /*CUDA_CHECK_RETURN(hipMallocManaged((void**)&(left_left_warp_census[dis_iter]),image_census_size));
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(left_center_warp_census[dis_iter]),image_census_size));
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(right_center_warp_census[dis_iter]),image_census_size));*/
        CUDA_CHECK_RETURN(hipMallocManaged((void**)&(right_right_warp_census[dis_iter]),image_census_size));
    }

    /*CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_left_center_census,image_census_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_center_center_census,image_census_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_center_center_census,image_census_size));*/
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_right_center_census,image_census_size));



    dim3 census_block_size;
    census_block_size.x = 32;
    census_block_size.y = 32;

    dim3 census_grid_size;
    census_grid_size.x = (NEW_MAX_DISPARITY)*((IMG_WIDTH+census_block_size.x-1) / census_block_size.x);
    census_grid_size.y = (IMG_HEIGHT+census_block_size.y-1) / census_block_size.y;

    N_CenterSymmetricCnesusKernelSM2<<<census_grid_size,census_block_size,0>>>(right_right_warp_data,
                                                                           right_right_warp_census);
    //CUDA_CHECK_RETURN(hipStreamSynchronize(w_stream1));
    /*N_CenterSymmetricCnesusKernelSM2<<<census_grid_size,census_block_size,0>>>(right_center_warp_data,right_right_warp_data,
                                                                           right_center_warp_census,right_right_warp_census);*/

    /*N_CenterSymmetricCnesusKernelSM2<<<census_grid_size,census_block_size,0>>>(right_right_warp_data,right_center_warp_data,
            right_right_warp_census,right_center_warp_census);*/
    //CUDA_CHECK_RETURN(hipStreamSynchronize(w_stream2));

    // Do center census
    dim3 n_block_size;
    n_block_size.x=32;
    n_block_size.y=32;

    dim3 n_grid_size;
    n_grid_size.x=(IMG_WIDTH+n_block_size.x-1) / n_block_size.x;
    n_grid_size.y=(IMG_HEIGHT+n_block_size.y-1) / n_block_size.y;
    CenterSymmetricCensusKernelSM2<<<n_grid_size, n_block_size,0>>>(right_right_center_data,
                                                                        right_right_center_census,IMG_HEIGHT, IMG_WIDTH);
    //CUDA_CHECK_RETURN(hipStreamSynchronize(w_stream3));
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    /*CenterSymmetricCensusKernelSM2<<<n_grid_size, n_block_size,0>>>(right_center_center_data, right_right_center_data,
            right_center_center_census, right_right_center_census, IMG_HEIGHT, IMG_WIDTH);*/
    /*CenterSymmetricCensusKernelSM2<<<n_grid_size, n_block_size,0>>>(right_right_center_data, right_center_center_data,
            right_right_center_census, right_center_center_census, IMG_HEIGHT, IMG_WIDTH);
    //CUDA_CHECK_RETURN(hipStreamSynchronize(w_stream4));
    CUDA_CHECK_RETURN(hipDeviceSynchronize());*/

    cv::Mat result_img = Process(right_right_warp_census,right_right_center_census,
                                           image_width,image_height,
                                           right_right_disparity
                                           );
    return result_img;

}

void TestCostCompute(cost_t **left_left_warp_census,cost_t *left_left_center_census,
                     float *left_left_disparity,
                     int image_width,int image_height,
                     uint32_t*& final_cost_volume
                     )
{
    int dis_num = NEW_MAX_DISPARITY;

    uint32_t max_cost =300;


    for(int image_y =0;image_y<image_height;image_y++){
        for(int image_x = 0;image_x<image_width;image_x++)
        {
            for(int dis_iter = 0;dis_iter<dis_num;dis_iter++) {
                int cost_index = (image_y * image_width + image_x) * dis_num + dis_iter;
                int image_index = (image_y * image_width) + image_x;
                if (image_x + left_left_disparity[dis_iter] >= image_width)
                    final_cost_volume[cost_index] = max_cost;
                else {
                    uint32_t answer = (left_left_center_census[image_index] ^
                                       (left_left_warp_census[dis_iter][image_index])); //Hamming Distance
                    uint32_t dist = 0;
                    while (answer) {
                        ++dist;
                        answer &= answer - 1;
                    }

                    final_cost_volume[cost_index] = dist;
                    //printf("%d::%d::%d\n",dis_iter,dist,final_cost_volume[cost_index]);

                }
            }
        }
    }

}
//for compute the cost volume that needed by cost aggregation
/*cv::Mat DepthComputeUtil::Process(cost_t **left_left_warp_census, cost_t *left_left_center_census,
                                         cost_t **left_center_warp_census, cost_t *left_center_center_census,
                                         cost_t **right_center_warp_census, cost_t *right_center_center_census,
                                         cost_t **right_right_warp_census, cost_t *right_right_center_census,
                                         int image_width,int image_height,
                                         StereoRecitified& stereoRecitified,
                                         float* left_left_disparity, float* left_center_disparity,
                                         float* right_center_disparity, float* right_right_disparity
                                         )*/
cv::Mat DepthComputeUtil::Process(
                                  cost_t **right_right_warp_census, cost_t *right_right_center_census,
                                  int image_width,int image_height,
                                  float* right_right_disparity
)
{

    hipStream_t n_stream1,n_stream2,n_stream3,n_stream4;
    CUDA_CHECK_RETURN(hipStreamCreate(&n_stream1));
    CUDA_CHECK_RETURN(hipStreamCreate(&n_stream2));
    CUDA_CHECK_RETURN(hipStreamCreate(&n_stream3));
    CUDA_CHECK_RETURN(hipStreamCreate(&n_stream4));

    uint32_t *final_cost_volume;
    int cost_size =NEW_MAX_DISPARITY*IMG_HEIGHT*IMG_WIDTH*sizeof(uint32_t);
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&final_cost_volume,cost_size));

    /*float *left_left_x,*left_left_y;
    float *left_center_x,*left_center_y;
    float *right_center_x,*right_center_y;
    float *right_right_x,*right_right_y;

    int map_size = image_width*image_height*sizeof(float);
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_left_x,map_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_left_y,map_size));

    CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_center_x,map_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&left_center_y,map_size));

    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_center_x,map_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_center_y,map_size));

    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_right_x,map_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void**)&right_right_y,map_size));

    StereoRecitified new_stereoRecitified;
    new_stereoRecitified.Preprocess(0);
    for(int img_x = 0;img_x<image_width;img_x++){
        for(int img_y = 0;img_y<image_height;img_y++){
            int img_index = img_x+img_y*image_width;
            new_stereoRecitified.GetCorrespondCoordinate(img_x,img_y,0,left_left_x[img_index],left_left_y[img_index]);
        }
    }
    new_stereoRecitified.Preprocess(1);
    for(int img_x = 0;img_x<image_width;img_x++){
        for(int img_y = 0;img_y<image_height;img_y++){
            int img_index = img_x+img_y*image_width;
            new_stereoRecitified.GetCorrespondCoordinate(img_x,img_y,1,left_center_x[img_index],left_center_y[img_index]);
        }
    }
    new_stereoRecitified.Preprocess(2);
    for(int img_x = 0;img_x<image_width;img_x++){
        for(int img_y = 0;img_y<image_height;img_y++){
            int img_index = img_x+img_y*image_width;
            new_stereoRecitified.GetCorrespondCoordinate(img_x,img_y,2,right_center_x[img_index],right_center_y[img_index]);
        }
    }
    new_stereoRecitified.Preprocess(3);

    for(int img_x = 0;img_x<image_width;img_x++){
        for(int img_y = 0;img_y<image_height;img_y++) {
            int img_index = img_x+img_y*image_width;
            new_stereoRecitified.GetCorrespondCoordinate(img_x,img_y,3,right_right_x[img_index],right_right_y[img_index]);
        }
    }
    for(int img_y = 0;img_y<image_height;img_y++){
        for (int img_x = 0; img_x < image_width; img_x++) {
            int img_index = img_x + img_y * image_width;

            //std::cout<<left_left_x[img_index]<<" "<<left_center_x[img_index]<<" "<<right_center_x[img_index]<<std::endl;
        }
    }*/
    //for(;;);
    /*for(int img_x = 0;img_x<image_width;img_x++){
        for(int img_y = 0;img_y<image_height;img_y++){
            int img_index = img_x+img_y*image_width;
            stereoRecitified.GetCorrespondCoordinate(img_x,img_y,0,left_left_x[img_index],left_left_y[img_index]);
            stereoRecitified.GetCorrespondCoordinate(img_x,img_y,1,left_center_x[img_index],left_center_y[img_index]);
            stereoRecitified.GetCorrespondCoordinate(img_x,img_y,2,right_center_x[img_index],right_center_y[img_index]);
            stereoRecitified.GetCorrespondCoordinate(img_x,img_y,3,right_right_x[img_index],right_right_y[img_index]);

        }
    }*/
    //TestCostCompute(left_left_warp_census,left_left_center_census,left_left_disparity,image_width,image_height,final_cost_volume);

    dim3 cost_grid_size;
    //  cost_grid_size.x = cols*16;
    cost_grid_size.x = image_width;
    cost_grid_size.y = image_height;

    dim3 cost_block_size;
    cost_block_size.x = 1;
    //cost_block_size=0;
    cost_block_size.y = NEW_MAX_DISPARITY;



    /*ComputeCostVolume<<<cost_grid_size, cost_block_size,0>>>(
            left_center_warp_census,left_center_center_census,
            left_left_warp_census,left_left_center_census,
            right_center_warp_census,right_center_center_census,
            right_right_warp_census,right_right_center_census,
            final_cost_volume,
            image_width,image_height,
            left_center_x,left_center_y,
            left_left_x,left_left_y,
            right_center_x,right_center_y,
            right_right_x,right_right_y,
            left_center_disparity,left_left_disparity,
            right_center_disparity,right_right_disparity
    );*/
    ComputeCostVolume<<<cost_grid_size, cost_block_size,0>>>(
                                                          right_right_warp_census,right_right_center_census,
                                                          final_cost_volume,
                                                          image_width,image_height,
                                                          right_right_disparity
                                                          );

    hipError_t  err = hipGetLastError();
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // hipEventRecord(stop, 0);
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }
    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    /*cv::Mat test_result_img(IMG_HEIGHT,IMG_WIDTH,CV_8UC1);
    std::cout<<"Process"<<std::endl;
    int sum_count = 0;
    for(int img_y =0;img_y<IMG_HEIGHT;img_y++){
        for(int img_x = 0;img_x<IMG_WIDTH;img_x++){
            test_result_img.at<uint8_t>(img_y,img_x) = 255;
            uint32_t max_cost = 1000;
            uint8_t r_index =200;
            int img_index = img_x+img_y*IMG_WIDTH;
            for(uint8_t dis_iter =0;dis_iter<NEW_MAX_DISPARITY;dis_iter++){
                if(final_cost_volume[img_index*NEW_MAX_DISPARITY+dis_iter]>300)
                    std::cout<<"Max_value::"<<final_cost_volume[img_index*NEW_MAX_DISPARITY+dis_iter]<<std::endl;
                if(final_cost_volume[img_index*NEW_MAX_DISPARITY+dis_iter]<max_cost) {
                    max_cost = final_cost_volume[img_index * NEW_MAX_DISPARITY + dis_iter];
                    r_index = dis_iter;

                }
            }
            //printf("%d::\n",r_index);

            sum_count++;
            //printf("%d\n",r_index);
            //if(r_index < 30)
            //    std::cout<<r_index<<" "<<max_cost<<" "<<left_left_x[img_index]<<" "<<left_left_y[img_index]<<std::endl;

            test_result_img.at<uint8_t>(img_y,img_x) = r_index;
        }
    }
    std::cout<<"sum count::"<<sum_count<<std::endl;
    return test_result_img;*/

    //for(int m =500*NEW_MAX_DISPARITY;m<501*NEW_MAX_DISPARITY;m++)
    //    std::cout<<"Cost::"<<int(final_cost_volume[m])<<std::endl;

    // Cost Aggregation

    //cost Aggregation Allocate

    uint32_t *new_d_L0;
    uint32_t *new_d_L1;
    uint32_t *new_d_L2;
    uint32_t *new_d_L3;
    uint32_t *new_d_L4;
    uint32_t *new_d_L5;
    uint32_t *new_d_L6;
    uint32_t *new_d_L7;

    uint8_t *d_disparity;

    const int l_cost_size = image_width*image_height*NEW_MAX_DISPARITY;

    CUDA_CHECK_RETURN(hipMallocManaged((void **)&new_d_L0, sizeof(uint32_t)*l_cost_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&new_d_L1, sizeof(uint32_t)*l_cost_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&new_d_L2, sizeof(uint32_t)*l_cost_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&new_d_L3, sizeof(uint32_t)*l_cost_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&new_d_L4, sizeof(uint32_t)*l_cost_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&new_d_L5, sizeof(uint32_t)*l_cost_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&new_d_L6, sizeof(uint32_t)*l_cost_size));
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&new_d_L7, sizeof(uint32_t)*l_cost_size));

    CUDA_CHECK_RETURN(hipMallocManaged((void **)&d_disparity, sizeof(uint8_t)*image_height*image_width));




    const int PIXELS_PER_BLOCK = COSTAGG_BLOCKSIZE/WARP_SIZE;//128/128
    const int PIXELS_PER_BLOCK_HORIZ = COSTAGG_BLOCKSIZE_HORIZ/WARP_SIZE;//128/128
    // std::cout<<"threads number::"<<COSTAGG_BLOCKSIZE_HORIZ<<std::endl;
    //std::cout<<"block number::"<<(rows+PIXELS_PER_BLOCK_HORIZ-1)/PIXELS_PER_BLOCK_HORIZ<<std::endl;
    debug_log("Calling Left to Right");
    const int rows = image_height;
    const int cols = image_width;

    const int size = image_height*image_width;
    const int size_cube_l = size*NEW_MAX_DISPARITY;
    N_CostAggregationKernelLeftToRight<<<(rows+PIXELS_PER_BLOCK_HORIZ-1)/PIXELS_PER_BLOCK_HORIZ, COSTAGG_BLOCKSIZE_HORIZ, 0, n_stream1>>>(final_cost_volume, new_d_L0, p1, p2, rows, cols, right_right_center_census, right_right_center_census, d_disparity, new_d_L0, new_d_L1, new_d_L2, new_d_L3, new_d_L4, new_d_L5, new_d_L6);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }
    debug_log("Calling Right to Left");
    N_CostAggregationKernelRightToLeft<<<(rows+PIXELS_PER_BLOCK_HORIZ-1)/PIXELS_PER_BLOCK_HORIZ, COSTAGG_BLOCKSIZE_HORIZ, 0, n_stream1>>>(final_cost_volume, new_d_L1, p1, p2, rows, cols, right_right_center_census, right_right_center_census, d_disparity, new_d_L0, new_d_L1, new_d_L2, new_d_L3, new_d_L4, new_d_L5, new_d_L6);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }
    debug_log("Calling Up to Down");
    N_CostAggregationKernelUpToDown<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, n_stream1>>>(final_cost_volume, new_d_L2, p1, p2, rows, cols, right_right_center_census, right_right_center_census, d_disparity, new_d_L0, new_d_L1, new_d_L2, new_d_L3, new_d_L4, new_d_L5, new_d_L6);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    debug_log("Calling Down to Up");
    std::cout<<COSTAGG_BLOCKSIZE<<std::endl;
    // for(;;);
    N_CostAggregationKernelDownToUp<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, n_stream1>>>(final_cost_volume, new_d_L3, p1, p2, rows, cols, right_right_center_census, right_right_center_census, d_disparity, new_d_L0, new_d_L1, new_d_L2, new_d_L3, new_d_L4, new_d_L5, new_d_L6);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }

#if PATH_AGGREGATION == 8
    N_CostAggregationKernelDiagonalDownUpLeftRight<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, n_stream1>>>(final_cost_volume, new_d_L4, p1, p2, rows, cols, right_right_center_census, right_right_center_census, d_disparity, new_d_L0, new_d_L1, new_d_L2, new_d_L3, new_d_L4, new_d_L5, new_d_L6);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }
    N_CostAggregationKernelDiagonalUpDownLeftRight<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, n_stream1>>>(final_cost_volume, new_d_L5, p1, p2, rows, cols, right_right_center_census, right_right_center_census, d_disparity, new_d_L0, new_d_L1, new_d_L2, new_d_L3, new_d_L4, new_d_L5, new_d_L6);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }

    N_CostAggregationKernelDiagonalDownUpRightLeft<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, n_stream1>>>(final_cost_volume, new_d_L6, p1, p2, rows, cols, right_right_center_census, right_right_center_census, d_disparity, new_d_L0, new_d_L1, new_d_L2, new_d_L3, new_d_L4, new_d_L5, new_d_L6);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }
    N_CostAggregationKernelDiagonalUpDownRightLeft<<<(cols+PIXELS_PER_BLOCK-1)/PIXELS_PER_BLOCK, COSTAGG_BLOCKSIZE, 0, n_stream1>>>(final_cost_volume, new_d_L7, p1, p2, rows, cols, right_right_center_census, right_right_center_census, d_disparity, new_d_L0, new_d_L1, new_d_L2, new_d_L3, new_d_L4, new_d_L5, new_d_L6);
    err = hipGetLastError();
    //hipEventRecord(stop, 0);
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }
#endif
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    debug_log("Calling Median Filter");
    //MedianFilter3x3<<<(size+NEW_MAX_DISPARITY-1)/NEW_MAX_DISPARITY, NEW_MAX_DISPARITY, 0, n_stream1>>>(d_disparity, d_disparity_filtered_uchar, rows, cols);

    uint8_t *d_disparity_filtered_uchar;
    CUDA_CHECK_RETURN(hipMallocManaged((void **)&d_disparity_filtered_uchar, sizeof(uint8_t)*image_width*image_height));

    MedianFilter3x3<<<(size+NEW_MAX_DISPARITY-1)/NEW_MAX_DISPARITY, NEW_MAX_DISPARITY, 0, n_stream1>>>(d_disparity, d_disparity_filtered_uchar, rows, cols);

    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error: %s %d\n", hipGetErrorString(err), err);
        exit(-1);
    }

    hipEventRecord(stop, 0);
    CUDA_CHECK_RETURN(hipDeviceSynchronize());
    float elapsed_mas;
    hipEventElapsedTime(&elapsed_mas, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    std::cout<<"time::"<<elapsed_mas<<std::endl;
    debug_log("Copying final disparity to CPU");
    //CUDA_CHECK_RETURN(hipMemcpy(h_disparity, d_disparity_filtered_uchar, sizeof(uint8_t)*size, hipMemcpyDeviceToHost));

    uint8_t *h_disparity;
    h_disparity = new uint8_t[image_height*image_width];
    hipMemcpy(h_disparity, d_disparity_filtered_uchar, sizeof(uint8_t)*size, hipMemcpyDeviceToHost);
    //hipMemcpy(h_disparity, d_disparity, sizeof(uint8_t)*size, hipMemcpyDeviceToHost);
    // CUDA_CHECK_RETURN(hipMemcpy(h_disparity, d_disparity, sizeof(uint8_t)*size, hipMemcpyDeviceToHost));

    //********aggregation end*************************************************
    /*for(int m =0;m<image_height*image_width;m++)
        std::cout<<int(h_disparity[m])<<std::endl;*/
    cv::Mat n_disparity_image(rows,cols,CV_8UC1,h_disparity);
    std::cout<<"************************************************"<<std::endl;
    //imwrite("1.jpg",n_disparity_image);
    //for(;;);
    return n_disparity_image;
}