#include "hip/hip_runtime.h"
#include "image_warp.h"
#include "configuration.h"


__device__ void ImageWarp(const uint8_t *center_im,uint8_t *n_image,float dx,float dy)
{
    //  const int img_x=blockIdx.x*blockDim.x+threadIdx.x;
    //  const int img_y=blockIdx.y*blockDim.y+threadIdx.y;
    //printf("begin\n");
    int img_x=2*threadIdx.x;
    const int img_y=blockIdx.x;


    float n_x=img_x-dx,n_y=img_y-dy;
    //get the orign x and the orign y
    //printf("size::%d",sizeof(n_image)/sizeof(n_image[0]));
    if(n_x<FLOAT_EPS||n_x>IMG_WIDTH-1||n_y<FLOAT_EPS||n_y>IMG_HEIGHT-1)
        return ;
    int tl_x=(int)n_x,tl_y=(int)n_y,tr_x=n_x+1,tr_y=tl_y;
    int bl_x=tl_x,bl_y=tl_y+1,br_x=bl_x+1,br_y=bl_y;

    uint8_t tl_color=center_im[tl_x+tl_y*IMG_WIDTH];
    uint8_t tr_color=center_im[tr_x+tr_y*IMG_WIDTH];
    uint8_t bl_color=center_im[bl_x+bl_y*IMG_WIDTH];
    uint8_t br_color=center_im[br_x+br_y*IMG_WIDTH];

    uint8_t t_color=(tr_x-n_x)*tl_color+(n_x-tl_x)*tr_color;
    uint8_t b_color=(br_x-n_x)*bl_color+(n_x-bl_x)*br_color;
    int r_d=(br_y-n_y)*t_color+(n_y-tr_y)*b_color;
    uint8_t r_color=(br_y-n_y)*t_color+(n_y-tr_y)*b_color;
    n_image[img_x+img_y*IMG_WIDTH]=r_color;
    //n_image[0]=r_d;

    //fix for big size
    img_x = 2*threadIdx.x+1;
    n_x=img_x-dx,n_y=img_y-dy;
    //get the orign x and the orign y
    //printf("size::%d",sizeof(n_image)/sizeof(n_image[0]));
    if(n_x<FLOAT_EPS||n_x>IMG_WIDTH-1||n_y<FLOAT_EPS||n_y>IMG_HEIGHT-1)
        return ;
    tl_x=(int)n_x,tl_y=(int)n_y,tr_x=n_x+1,tr_y=tl_y;
    bl_x=tl_x,bl_y=tl_y+1,br_x=bl_x+1,br_y=bl_y;

    tl_color=center_im[tl_x+tl_y*IMG_WIDTH];
    tr_color=center_im[tr_x+tr_y*IMG_WIDTH];
    bl_color=center_im[bl_x+bl_y*IMG_WIDTH];
    br_color=center_im[br_x+br_y*IMG_WIDTH];

    t_color=(tr_x-n_x)*tl_color+(n_x-tl_x)*tr_color;
    b_color=(br_x-n_x)*bl_color+(n_x-bl_x)*br_color;
    r_d=(br_y-n_y)*t_color+(n_y-tr_y)*b_color;
    r_color=(br_y-n_y)*t_color+(n_y-tr_y)*b_color;
    n_image[img_x+img_y*IMG_WIDTH]=r_color;

}
// Do Image Shift
__global__ void Image_Shift(uint8_t *origin_image, uint8_t **warp_image,float *dis_list,
                            int px, int py)
{
    const int b_y=blockIdx.y;
    int image_n=b_y;

    int dis_n =  b_y;//current dis_label
    float now_dis = dis_list[dis_n];

    float d_x,d_y;
    d_y=py*now_dis,d_x=px*now_dis;
    ImageWarp(origin_image,warp_image[dis_n],d_x,d_y);
}

__device__ uint32_t CostCompute(float map_center_x, float map_center_y, int image_width, int image_height,
                      cost_t *center_census, cost_t **stereo_census, int dis_index){

    //printf("matrix::%f::%f\n",map_center_x,map_center_y);
    int left_map_x = int(map_center_x);
    int bottom_map_y = int(map_center_y);

    int right_map_x = left_map_x+1;
    int top_map_y = bottom_map_y+1;

    float right_map_x_dis = map_center_x-left_map_x;
    float left_map_x_dis = 1-right_map_x_dis;

    float top_map_y_dis = map_center_y-bottom_map_y;
    //printf("%f:::::%d\n",map_center_y,bottom_map_y);
    float bottom_map_y_dis = 1-top_map_y_dis;
    //float sum_cost = 0;
    int max_cost = 300;
    int t_l_cost=max_cost, t_r_cost=max_cost,b_l_cost=max_cost,b_r_cost=max_cost;
    int census_index;
    if(top_map_y<image_height){
        census_index = top_map_y * image_width+ left_map_x;
        t_l_cost = popcount(center_census[census_index]^stereo_census[dis_index][census_index]);
        if(right_map_x<image_width) {
            census_index = top_map_y * image_width + right_map_x;
            t_r_cost = popcount(center_census[census_index]^stereo_census[dis_index][census_index]);
        }
    }

    census_index = bottom_map_y * image_width+ left_map_x;
    b_l_cost = popcount(center_census[census_index]^stereo_census[dis_index][census_index]);
    if(right_map_x<image_width) {
        census_index = bottom_map_y * image_width + right_map_x;
        b_r_cost = popcount(center_census[census_index]^stereo_census[dis_index][census_index]);
    }

    //printf("%f::%f::%f::%f\n",top_map_y_dis,right_map_x_dis,left_map_x_dis,bottom_map_y_dis);
    float sum_cost = top_map_y_dis*(right_map_x_dis*t_r_cost+left_map_x_dis*t_l_cost)+bottom_map_y_dis*(right_map_x_dis*b_r_cost+left_map_x_dis*b_l_cost);

    uint32_t result_cost = (uint32_t)sum_cost;
    //printf("%f::%d\n",sum_cost,result_cost);
    return result_cost;

}
/*__global__ void ComputeCostVolume(cost_t **left_left_census, cost_t *left_left_center_census,
                                  cost_t **left_center_census, cost_t *left_center_center_census,
                                  cost_t **right_center_census, cost_t *right_center_center_census,
                                  cost_t **right_right_census, cost_t *right_right_center_census,
                                  uint32_t *final_cost_volume,
                                  int image_width, int image_height,
                                  float *left_left_x,float *left_left_y,
                                  float *left_center_x,float *left_center_y,
                                  float *right_center_x,float *right_center_y,
                                  float *right_right_x,float *right_right_y,
                                  float *left_left_disparity, float *left_center_disparity,
                                  float *right_center_disparity, float *right_right_disparity

)*/
__global__ void ComputeCostVolume(cost_t **right_right_census, cost_t *right_right_center_census,
                                  uint32_t *final_cost_volume,
                                  int image_width, int image_height,
                                  float *right_right_disparity

)
{
    __shared__ uint32_t costs[32];
    int b_x = blockIdx.x;
    int b_y = blockIdx.y;
    int t_x = threadIdx.x;
    int t_y = threadIdx.y;
    int image_num=t_x;
    int dis_n = (b_x/image_width)*32+t_y;
    int t_num = t_y;
    int image_w = b_x%image_width;
    int image_h = b_y;
    uint32_t n_result = 0;
    int image_p=image_num;
    const int image_mount = 1;
    //int image_count=image_num%SELECT_IMAGE_NUM;
    //image_count = IMAGE_NUMBER-1-image_count;
    //int t_pixel_count=(dis_n)*IMG_WIDTH*IMG_HEIGHT+image_h*IMG_WIDTH+image_w;

    int center_index = image_h*image_width+image_w;

    cost_t left_left_center_cost,left_center_center_cost;
    cost_t right_center_center_cost,right_right_center_cost;


    int census_image_num = dis_n;

    //int map_center_x,map_center_y;
    //t_pixel_count= dis_num*IMG_WIDTH*IMG_HEIGHT+image_h*IMG_WIDTH+image_w;
    const uint32_t max_cost =300;

    int census_index;
    float map_center_x,map_center_y;

    float now_dis,new_map_x;

    int left_map_x,right_map_x;
    float left_map_x_dis,right_map_x_dis;
    int top_map_y,bottom_map_y;
    float top_map_y_dis, bottom_map_y_dis;
    int left_bottom_cost,right_bottom_cost,left_top_cost,right_top_cost;

    {
        // right right pair
        /*map_center_x = int(right_right_x[center_index]);
        map_center_y = int(right_right_y[center_index]);
        census_index = map_center_y*image_width+map_center_x;

        now_dis = right_right_disparity[dis_n];
        new_map_x = map_center_x-now_dis;*/
        new_map_x = image_w-now_dis;
        census_index = image_h*image_width+image_w;
        if(image_w>=image_width||image_h>=image_height||new_map_x<0.0f)
            final_cost_volume[(image_h*IMG_WIDTH+image_w)*NEW_MAX_DISPARITY+dis_n]=max_cost;
            //costs[t_num*image_mount+image_num] = max_cost;
        else{
            right_right_center_cost = right_right_center_census[census_index];
            final_cost_volume[(image_h*IMG_WIDTH+image_w)*NEW_MAX_DISPARITY+dis_n] = popcount(
                    right_right_center_cost ^ (right_right_census[census_image_num][census_index]));
            //costs[t_num * image_mount + image_num]

        }
    }
    /*__syncthreads();
    //if(image_p==3)
    {

        int e_index=(t_num+1)*image_mount,b_index=t_num*image_mount;
        uint32_t s_result=0;
        //s_result=costs[b_index+2]+costs[b_index];
        //s_result=costs[b_index+2];
        //s_result=costs[b_index+2]+costs[b_index+1]+costs[b_index];

        //printf("%d\n",s_result);
        final_cost_volume[(image_h*IMG_WIDTH+image_w)*NEW_MAX_DISPARITY+dis_n]=s_result;
    }*/
    /*int b_index,e_index,h_index;
    if(image_num%4==0)
    {
        h_index=b_index=t_num*SUM_IMAGE_NUM+image_num,e_index=t_num*SUM_IMAGE_NUM+(image_num/4+1)*4;
        uint32_t s_result=0;
        for(;b_index<e_index;b_index++)
        {
            s_result+=costs[b_index];
        }
        costs[h_index]=s_result;

    }
    __syncthreads();
    if(image_num==15)
    {
        e_index=(t_num+1)*SUM_IMAGE_NUM,b_index=t_num*SUM_IMAGE_NUM;
        uint32_t s_result=0;
        for(;b_index<e_index;b_index+=4)
        {
            s_result+=costs[b_index];
        }
                //printf("done\n");
        d_cost[(image_h*IMG_WIDTH+image_w)*NEW_MAX_DISPARITY+dis_n]=s_result;
    }*/
}