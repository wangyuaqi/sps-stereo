#include "hip/hip_runtime.h"
/**
    This file is part of sgm. (https://github.com/dhernandez0/sgm).

    Copyright (c) 2016 Daniel Hernandez Juarez.

    sgm is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    sgm is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with sgm.  If not, see <http://www.gnu.org/licenses/>.

**/

#include "hamming_cost.h"

//d_transform0, d_transform1, d_cost, rows, cols
__global__ void
HammingDistanceCostKernel (  const cost_t *d_transform0, const cost_t *d_transform1,
		uint8_t *d_cost, const int rows, const int cols ) {
	//const int Dmax=   blockDim.x;  // Dmax is CTA size
	const int y=      blockIdx.x;  // y is CTA Identifier
	const int THRid = threadIdx.x; // THRid is Thread Identifier
   // printf("blockId::%d\n",y);
    __shared__ cost_t SharedMatch[MAX_DISPARITY];
    __shared__ cost_t SharedBase [MAX_DISPARITY/2];

    //printf("threadId::%d\n",THRid);
    SharedMatch [MAX_DISPARITY/2+THRid] = d_transform1[y*cols+0];  // init position

    int n_iter = 2*cols/MAX_DISPARITY;
	for (int ix=0; ix<n_iter; ix++) {
        const int x = ix*MAX_DISPARITY/2;
        SharedMatch [THRid]      = SharedMatch [THRid + MAX_DISPARITY/2];
        SharedMatch [THRid+MAX_DISPARITY/2] = d_transform1 [y*cols+x+THRid];
		SharedBase  [THRid]      = d_transform0 [y*cols+x+THRid];

		__syncthreads();
        for (int i=0; i<MAX_DISPARITY/2; i++) {
			const cost_t base  = SharedBase [i];
            const cost_t match = SharedMatch[(MAX_DISPARITY/2-1-THRid)+1+i];
            d_cost[(y*cols+x+i)*MAX_DISPARITY+THRid] = 30;
            //*************
            d_cost[(y*cols+x+i)*MAX_DISPARITY+MAX_DISPARITY/2+THRid] = popcount( base ^ match );
		}
		__syncthreads();
	}
	// For images with cols not multiples of MAX_DISPARITY
    const int x = MAX_DISPARITY/2*(2*cols/MAX_DISPARITY);
	const int left = cols-x;
	if(left > 0) {
        SharedMatch [THRid]      = SharedMatch [THRid + MAX_DISPARITY/2];
		if(THRid < left) {
            SharedMatch [THRid+MAX_DISPARITY/2] = d_transform1 [y*cols+x+THRid];
			SharedBase  [THRid]      = d_transform0 [y*cols+x+THRid];
		}

		__syncthreads();
		for (int i=0; i<left; i++) {
			const cost_t base  = SharedBase [i];
            const cost_t match = SharedMatch[(MAX_DISPARITY/2-1-THRid)+1+i];
            d_cost[(y*cols+x+i)*MAX_DISPARITY+THRid] = 300;
            //************
            d_cost[(y*cols+x+i)*MAX_DISPARITY+THRid+MAX_DISPARITY/2] = popcount( base ^ match );

		}
		__syncthreads();
	}
}


//**********
__global__ void
HammingDistanceCostKernel_Z (  const cost_t *d_transform0, const cost_t *d_transform1,
        uint8_t *d_cost, const int rows, const int cols ) {
    //const int Dmax=   blockDim.x;  // Dmax is CTA size
    const int y=      blockIdx.x;  // y is CTA Identifier
    const int THRid = threadIdx.x; // THRid is Thread Identifier
    //printf("blockId::%d\n",y);
    __shared__ cost_t SharedMatch_Z[MAX_DISPARITY];
    __shared__ cost_t SharedBase_Z [MAX_DISPARITY/2];

    //printf("new hamming distance%u::\n",d_transform0[0]);
    //printf("threadId::%d\n",THRid);
    SharedMatch_Z [THRid] = d_transform1[y*cols+cols-1];  // init position
    //int min_pop=100;
    int n_iter = 2*cols/MAX_DISPARITY;
    for (int ix=n_iter; ix>0; ix--) {
        const int x = ix*MAX_DISPARITY/2;
        SharedMatch_Z [THRid+MAX_DISPARITY/2]= SharedMatch_Z [THRid];
        SharedMatch_Z [MAX_DISPARITY/2-1-THRid] = d_transform1 [y*cols+x-THRid-1];
        SharedBase_Z  [MAX_DISPARITY/2-1-THRid]      = d_transform0 [y*cols+x-THRid-1];

        __syncthreads();
        for (int i=0; i<MAX_DISPARITY/2; i++) {
            const cost_t base  = SharedBase_Z [i];
            //const cost_t match = SharedMatch_Z[(MAX_DISPARITY/2-1-THRid)+1+i];
            const cost_t match = SharedMatch_Z[i+THRid];
            //if(min_pop>popcount( base ^ match ))
                //min_pop=popcount( base ^ match );
            d_cost[(y*cols+x-MAX_DISPARITY/2+i)*MAX_DISPARITY+MAX_DISPARITY/2-1-THRid] = popcount( base ^ match );
            //*************
            //d_cost[(y*cols+x-MAX_DISPARITY/2+i)*MAX_DISPARITY+MAX_DISPARITY-THRid] = 40;
            //d_cost[(y*cols+x-MAX_DISPARITY/2+i)*MAX_DISPARITY+MAX_DISPARITY/2+THRid] = 40;
        }
        __syncthreads();
    }
   // printf("min_pop::%d\n",min_pop);
    // For images with cols not multiples of MAX_DISPARITY
    const int x = MAX_DISPARITY/2*(2*cols/MAX_DISPARITY);
    //const int left = cols-x;
    const int left=cols-x;
    /*if(left > 0) {
        SharedMatch_Z [THRid]      = SharedMatch_Z [THRid + MAX_DISPARITY/2];
        if(THRid < left) {
            SharedMatch_Z [THRid+MAX_DISPARITY/2] = d_transform1 [y*cols+x-THRid];
            SharedBase_Z  [THRid]      = d_transform0 [y*cols+x-THRid];
        }

        __syncthreads();
        for (int i=0; i<left; i++) {
            const cost_t base  = SharedBase_Z [i];
            const cost_t match = SharedMatch_Z[(MAX_DISPARITY/2-1-THRid)+1+i];
            d_cost[(y*cols+x+i)*MAX_DISPARITY+THRid] = 300;
            //************
            d_cost[(y*cols+x+i)*MAX_DISPARITY+THRid+MAX_DISPARITY/2] = popcount( base ^ match );
        }
        __syncthreads();
    }*/
    if(left>0)
    {
        SharedMatch_Z [THRid]      = SharedMatch_Z [THRid + MAX_DISPARITY/2];
        if(THRid < left) {
            SharedMatch_Z [THRid+MAX_DISPARITY/2] = d_transform1 [y*cols+x-THRid];
            SharedBase_Z  [THRid]      = d_transform0 [y*cols+x-THRid];
        }

        __syncthreads();
        for (int i=0; i<left; i++) {
            const cost_t base  = SharedBase_Z [i];
            const cost_t match = SharedMatch_Z[(MAX_DISPARITY/2-1-THRid)+1+i];
            //d_cost[(y*cols+x+i)*MAX_DISPARITY+THRid] = 300;
            //************
            d_cost[(y*cols+x+i)*MAX_DISPARITY+THRid+MAX_DISPARITY/2] = popcount( base ^ match );
        }
        __syncthreads();
    }
}
