#include "hip/hip_runtime.h"
#include"n_hamming_cost.h"
/*__global__ void
N_HammingDistanceCostKernel (  cost_t *l_transform1,cost_t *l_transform2,cost_t *l_transform3,cost_t *l_transform4,
                               cost_t *r_transform1 ,cost_t *r_transform2 ,cost_t *r_transform3 ,cost_t *r_transform4 ,
                               const cost_t* center_transform,
                               cost_t *t_transform1,cost_t *t_transform2,cost_t *t_transform3,cost_t *t_transform4,
                               cost_t *b_transform1,cost_t *b_transform2,cost_t *b_transform3,cost_t *b_transform4,
                               uint8_t *d_cost, const int rows, const int cols,const int image_number,const int sum_disparity)*/
__global__ void
N_HammingDistanceCostKernel1 (  cost_t *l_transform1,
                                cost_t *r_transform1 ,
                               const cost_t* center_transform,
                               cost_t *t_transform1,
                               cost_t *b_transform1,
                               uint32_t *d_cost, const int rows, const int cols,const int image_number,const int sum_disparity)
{
    const int x=   blockIdx.x;  // the center image pixel x
    const int y=   blockIdx.y;  // the center image pixel y
    const int THRid = threadIdx.x; // represend the cost label
    //l_transfgorm b_transform is right top
    //cost_t center_tran=center_transform[y*cols+x];
    //float disparity=0.05*THRid*image_number;

    float disparity=0.1*(THRid+1)*image_number;
    int l_disparity=int(disparity);
    //float l_r=float(disparity-l_disparity);
    int r_disparity=(disparity-l_disparity>0)?(l_disparity+1):(l_disparity);
    float dif=disparity-l_disparity;
   // int result=0;
    //int n_result=0;
    int l_dif=int((1.0f-dif)*10.0f+0.1f);
    int r_dif=int(dif*10.0f+0.1f);
    const cost_t center_cost=center_transform[x+y*cols];
    const cost_t r=~center_cost;
    //printf("%d::%d\n",l_dif,r_dif);
   // printf("%d\n",popcount(center_cost^r));
    //cost_t r=0;
    /*cost_t l_r_cost=(x+r_disparity)<cols?l_transform1[x+r_disparity+y*cols]:l_transform1[cols-1+y*cols];
    cost_t l_l_cost=(x+l_disparity)<cols?l_transform1[x+l_disparity+y*cols]:l_transform1[cols-1+y*cols];

    cost_t r_r_cost=(x-r_disparity)>=0?r_transform1[x-r_disparity+y*cols]:r_transform1[0+y*cols];
    cost_t r_l_cost=(x-l_disparity)>=0?r_transform1[x-l_disparity+y*cols]:r_transform1[0+y*cols];

    cost_t b_r_cost=(y+r_disparity)<rows?b_transform1[x+(y+r_disparity)*cols]:b_transform1[x+(rows-1)*cols];
    cost_t b_l_cost=(y+l_disparity)<rows?b_transform1[x+(y+l_disparity)*cols]:b_transform1[x+(rows-1)*cols];

    cost_t t_r_cost=(y-r_disparity)>=0?t_transform1[x+(y-r_disparity)*cols]:t_transform1[x+0*cols];
    cost_t t_l_cost=(y-l_disparity)>=0?t_transform1[x+(y-l_disparity)*cols]:t_transform1[x+0*cols];*/

    const cost_t l_r_cost=(x+r_disparity)<cols?l_transform1[x+r_disparity+y*cols]:r;
    const cost_t l_l_cost=(x+l_disparity)<cols?l_transform1[x+l_disparity+y*cols]:r;

    const cost_t r_r_cost=(x-r_disparity)>=0?r_transform1[x-r_disparity+y*cols]:r;
    const cost_t r_l_cost=(x-l_disparity)>=0?r_transform1[x-l_disparity+y*cols]:r;

    const cost_t b_r_cost=(y+r_disparity)<rows?b_transform1[x+(y+r_disparity)*cols]:r;
    const cost_t b_l_cost=(y+l_disparity)<rows?b_transform1[x+(y+l_disparity)*cols]:r;

    const cost_t t_r_cost=(y-r_disparity)>=0?t_transform1[x+(y-r_disparity)*cols]:r;
    const cost_t t_l_cost=(y-l_disparity)>=0?t_transform1[x+(y-l_disparity)*cols]:r;


    const int result=(l_dif*((popcount(center_cost^l_l_cost)+popcount(center_cost^r_l_cost)+popcount(center_cost^b_l_cost)+popcount(center_cost^t_l_cost)))
     +r_dif*((popcount(center_cost^l_r_cost)+popcount(center_cost^r_r_cost)+popcount(center_cost^b_r_cost)+popcount(center_cost^t_r_cost))));
    /*const int result=(l_dif*(popcount(center_cost^b_l_cost)+popcount(center_cost^t_l_cost)))
             +r_dif*(+popcount(center_cost^b_r_cost)+popcount(center_cost^t_r_cost));*/
    /*const int result=(l_dif*(popcount(center_cost^l_l_cost)+popcount(center_cost^r_l_cost)))
             +r_dif*(+popcount(center_cost^r_r_cost)+popcount(center_cost^r_r_cost));*/

    d_cost[(x+y*cols)*NEW_MAX_DISPARITY+NEW_MAX_DISPARITY/2+THRid]+=(result);
    //if(d_cost[(x+y*cols)*NEW_MAX_DISPARITY+NEW_MAX_DISPARITY/2-1-THRid]==0)
     // printf("%u::result::%d::old_id::%d\n",d_cost[(x+y*cols)*NEW_MAX_DISPARITY+NEW_MAX_DISPARITY/2-1-THRid],result,NEW_MAX_DISPARITY/2+THRid);
    //***********positive d_cost finish******************************
    //***********negative d_cost begin*******************************
    /*l_r_cost=(x-r_disparity)>=0?l_transform1[x-r_disparity+y*cols]:l_transform1[0+y*cols];
    l_l_cost=(x-l_disparity)>=0?l_transform1[x-l_disparity+y*cols]:l_transform1[0+y*cols];

    r_r_cost=(x+r_disparity)<cols?r_transform1[x+r_disparity+y*cols]:r_transform1[cols-1+y*cols];
    r_l_cost=(x+l_disparity)<cols?r_transform1[x+l_disparity+y*cols]:r_transform1[cols-1+y*cols];

    b_r_cost=(y-r_disparity)>=0?b_transform1[x+(y-r_disparity)*cols]:b_transform1[x+0*cols];
    b_l_cost=(y-l_disparity)>=0?b_transform1[x+(y-l_disparity)*cols]:b_transform1[x+0*cols];

    t_r_cost=(y+r_disparity)<rows?t_transform1[x+(y+r_disparity)*cols]:t_transform1[x+(rows-1)*cols];
    t_l_cost=(y+l_disparity)<rows?t_transform1[x+(y+l_disparity)*cols]:t_transform1[x+(rows-1)*cols];*/


    const cost_t l_r_cost_2=(x-r_disparity)>=0?l_transform1[x-r_disparity+y*cols]:r;
    const cost_t l_l_cost_2=(x-l_disparity)>=0?l_transform1[x-l_disparity+y*cols]:r;

    const cost_t r_r_cost_2=(x+r_disparity)<cols?r_transform1[x+r_disparity+y*cols]:r;
    const cost_t r_l_cost_2=(x+l_disparity)<cols?r_transform1[x+l_disparity+y*cols]:r;

    const cost_t b_r_cost_2=(y-r_disparity)>=0?b_transform1[x+(y-r_disparity)*cols]:r;
    const cost_t b_l_cost_2=(y-l_disparity)>=0?b_transform1[x+(y-l_disparity)*cols]:r;

    const cost_t t_r_cost_2=(y+r_disparity)<rows?t_transform1[x+(y+r_disparity)*cols]:r;
    const cost_t t_l_cost_2=(y+l_disparity)<rows?t_transform1[x+(y+l_disparity)*cols]:r;

    //int f1=(1.0-dif)*100*((popcount(center_cost^l_l_cost)+popcount(center_cost^r_l_cost)+popcount(center_cost^b_l_cost)+popcount(center_cost^t_l_cost))/4);
    //int f2=dif*100*((popcount(center_cost^l_r_cost)+popcount(center_cost^r_r_cost)+popcount(center_cost^b_r_cost)+popcount(center_cost^t_r_cost))/4);
    const int n_result=(l_dif*((popcount(center_cost^l_l_cost_2)+popcount(center_cost^r_l_cost_2)+popcount(center_cost^b_l_cost_2)+popcount(center_cost^t_l_cost_2)))
             +r_dif*((popcount(center_cost^l_r_cost_2)+popcount(center_cost^r_r_cost_2)+popcount(center_cost^b_r_cost_2)+popcount(center_cost^t_r_cost_2))));
    /*const int n_result=(l_dif*(popcount(center_cost^b_l_cost_2)+popcount(center_cost^t_l_cost_2)))
             +r_dif*(+popcount(center_cost^b_r_cost_2)+popcount(center_cost^t_r_cost_2));*/
   /* const int n_result=(l_dif*(popcount(center_cost^l_l_cost)+popcount(center_cost^r_l_cost)))
             +r_dif*(+popcount(center_cost^r_r_cost)+popcount(center_cost^r_r_cost));*/

    d_cost[(x+y*cols)*NEW_MAX_DISPARITY+NEW_MAX_DISPARITY/2-1-THRid]+=(n_result);
    //if(n_result==0)
      //printf("%u::id::%d\n",d_cost[(x+y*cols)*NEW_MAX_DISPARITY+NEW_MAX_DISPARITY/2+THRid],NEW_MAX_DISPARITY/2+THRid);*/

    //if(d_cost[(x+y*cols)*NEW_MAX_DISPARITY+NEW_MAX_DISPARITY/2+THRid]!=n_result)
    //printf("cost:::%u::%d\n",d_cost[(x+y*cols)*NEW_MAX_DISPARITY+NEW_MAX_DISPARITY/2+THRid],n_result);
}

//****************Warp Disparity***********
__global__ void
W_N_HammingDistanceKernel(cost_t* center_transform,
                          cost_t* l_c_transform,cost_t *r_c_transform,cost_t *b_c_transform,cost_t *t_c_transform,
                          cost_t *top_left_c_transform,cost_t *top_right_c_transform, cost_t *bottom_left_c_transform,cost_t *bottom_right_c_transform,
                          uint8_t *c_pic,uint8_t *l_pic,uint8_t *r_pic,uint8_t *b_pic,uint8_t *t_pic,
                          uint8_t *top_left_pic,uint8_t *top_righth_pic,uint8_t *bottom_left_pic, uint8_t *bottom_right_pic,
                          uint32_t *d_cost,
                          float* right_f_vec,float* right_baseline_vec,
                          float* left_f_vec,float* left_baseline_vec
                          )
{
    // __shared__ uint32_t cost_z[NEW_MAX_DISPARITY*16];
    /*const int b_x=blockIdx.x;
    const int by=blockIdx.y;
    const int dis_n=threadIdx.x;

    const int bx=b_x%IMG_WIDTH;
    const int img_n=(b_x/IMG_WIDTH)/4;
    const int img_p=(b_x/IMG_WIDTH)%4;
   // const int img_n=(b_x/IMG_WIDTH);
    uint32_t n_result=0;
    int dis_count,pixel_count,t_pixel_count;
    pixel_count=by*IMG_WIDTH+bx;
    dis_count=img_n*NEW_MAX_DISPARITY+dis_n;
    t_pixel_count=dis_count*(IMG_HEIGHT*IMG_WIDTH)+pixel_count;
    cost_t center_cost=center_transform[pixel_count];//get the center cost
    int cost_index=pixel_count*NEW_MAX_DISPARITY+dis_n;
    if(img_p==0)
      n_result=popcount(center_cost^l_c_transform[t_pixel_count]);
    else if(img_p==1)
      n_result=popcount(center_cost^r_c_transform[t_pixel_count]);
    else if(img_p==2)
      n_result=popcount(center_cost^t_c_transform[t_pixel_count]);
    else
      n_result=popcount(center_cost^b_c_transform[t_pixel_count]);
     d_cost[cost_index]+=(n_result);*/
    __shared__ uint32_t costs[SUM_IMAGE_NUM*32];
    __shared__ uint32_t new_costs[SUM_IMAGE_NUM*32];
    int b_x=blockIdx.x;
    int b_y=blockIdx.y;
    int t_x=threadIdx.x;
    int t_y=threadIdx.y;
    int image_num=t_x;
    int dis_n=(b_x/IMG_WIDTH)*32+t_y;
    int t_num=t_y;
    int image_w=b_x%IMG_WIDTH;
    int image_h=b_y;
    uint32_t n_result=0;
    int image_p=image_num/SELECT_IMAGE_NUM;
    int image_count=image_num%SELECT_IMAGE_NUM;
    //image_count = IMAGE_NUMBER-1-image_count;
    int t_pixel_count=(dis_n+image_count*NEW_MAX_DISPARITY)*IMG_WIDTH*IMG_HEIGHT+image_h*IMG_WIDTH+image_w;
    cost_t center_cost=center_transform[image_h*IMG_WIDTH+image_w];
    //if(image_num%4!=0&&image_num%4!=1&&image_num%4!=2)

    float now_dis;
    float now_depth = dis_n*(END_DEPTH-BEGIN_DEPTH)/float(NEW_MAX_DISPARITY)+BEGIN_DEPTH;
    //printf("::%f\n",left_baseline_vec[1]);
    //printf("%f\n",now_depth);
    int dis_num;
    if(image_p==0)
    {
        //costs[t_num*SUM_IMAGE_NUM+image_num]=popcount(center_cost^l_c_transform[t_pixel_count]);
        //new_costs[t_num*SUM_IMAGE_NUM+image_num]=popcount(center_cost^top_left_c_transform[t_pixel_count]);
        //printf("::%f\n",left_baseline_vec[0]);
        now_dis = left_baseline_vec[image_count]*left_f_vec[0]/now_depth;
        //printf("::%f\n",now_dis);
        if(now_dis>END_DIS)
            dis_num = NEW_MAX_DISPARITY-1;
        else
            dis_num = (now_dis-BEGIN_DIS)/((END_DIS-BEGIN_DIS)/float(NEW_MAX_DISPARITY));
        t_pixel_count= (dis_num+image_count*NEW_MAX_DISPARITY)*IMG_WIDTH*IMG_HEIGHT+image_h*IMG_WIDTH+image_w;
        costs[t_num*SUM_IMAGE_NUM+image_num]=popcount(center_cost^l_c_transform[t_pixel_count]);

        new_costs[t_num*SUM_IMAGE_NUM+image_num]=0;
    }
    else if(image_p==1)
    {
        now_dis = right_baseline_vec[image_count]*left_f_vec[0]/now_depth;

        if(now_dis>END_DIS)
            dis_num = NEW_MAX_DISPARITY-1;
        else
            dis_num = (now_dis-BEGIN_DIS)/((END_DIS-BEGIN_DIS)/float(NEW_MAX_DISPARITY));
        //if(now_dis<BEGIN_DIS)
        //printf("::%d\n",dis_num);
        t_pixel_count=(dis_num+image_count*NEW_MAX_DISPARITY)*IMG_WIDTH*IMG_HEIGHT+image_h*IMG_WIDTH+image_w;
        costs[t_num*SUM_IMAGE_NUM+image_num]=popcount(center_cost^r_c_transform[t_pixel_count]);
        /*if(image_w>=dis_n*WARP_DIS){
            costs[t_num*SUM_IMAGE_NUM+image_num]=popcount(center_cost^r_c_transform[t_pixel_count]);
        }
        else {
             costs[t_num*SUM_IMAGE_NUM+image_num]=300;
        }*/
        //new_costs[t_num*SUM_IMAGE_NUM+image_num]=popcount(center_cost^top_right_c_transform[t_pixel_count]);
        //costs[t_num*SUM_IMAGE_NUM+image_num] = 0;
        new_costs[t_num*SUM_IMAGE_NUM+image_num] = 0;
    }
    else if(image_p==2)
    {
        //printf("::%f\n",right_baseline_vec[0]);
        //costs[t_num*SUM_IMAGE_NUM+image_num]=popcount(center_cost^t_c_transform[t_pixel_count]);
        //new_costs[t_num*SUM_IMAGE_NUM+image_num]=popcount(center_cost^bottom_left_c_transform[t_pixel_count]);
        costs[t_num*SUM_IMAGE_NUM+image_num]=0;
        new_costs[t_num*SUM_IMAGE_NUM+image_num]=0;
    }
    else
    {
        //printf("::%f\n",right_baseline_vec[0]);
        //costs[t_num*SUM_IMAGE_NUM+image_num]=popcount(center_cost^b_c_transform[t_pixel_count]);
        //new_costs[t_num*SUM_IMAGE_NUM+image_num]=popcount(center_cost^bottom_right_c_transform[t_pixel_count]);
        costs[t_num*SUM_IMAGE_NUM+image_num]=0;
        new_costs[t_num*SUM_IMAGE_NUM+image_num]=0;
    }
    __syncthreads();
    if(image_num==4*SELECT_IMAGE_NUM-1)
    {
        //
        int e_index=(t_num+1)*SUM_IMAGE_NUM,b_index=t_num*SUM_IMAGE_NUM;
        uint32_t s_result=0;
        for(;b_index<e_index;b_index++)
        {
            //if((b_index%4)!=0&&(b_index%4!=1)&&(b_index%4!=2))
            s_result+=costs[b_index];
               // s_result+=new_costs[b_index];
        }
        //s_result = costs[e_index-1];
        //printf("done\n");
        d_cost[(image_h*IMG_WIDTH+image_w)*NEW_MAX_DISPARITY+dis_n]=s_result;
    }
    /*int b_index,e_index,h_index;
    if(image_num%4==0)
    {
        h_index=b_index=t_num*SUM_IMAGE_NUM+image_num,e_index=t_num*SUM_IMAGE_NUM+(image_num/4+1)*4;
        uint32_t s_result=0;
        for(;b_index<e_index;b_index++)
        {
            s_result+=costs[b_index];
        }
        costs[h_index]=s_result;

    }
    __syncthreads();
    if(image_num==15)
    {
        e_index=(t_num+1)*SUM_IMAGE_NUM,b_index=t_num*SUM_IMAGE_NUM;
        uint32_t s_result=0;
        for(;b_index<e_index;b_index+=4)
        {
            s_result+=costs[b_index];
        }
                //printf("done\n");
        d_cost[(image_h*IMG_WIDTH+image_w)*NEW_MAX_DISPARITY+dis_n]=s_result;
    }*/
}

__global__ void
N_N_HammingDistanceKernel(cost_t* center_transform,
                          cost_t* l_c_transform,
                         // uint8_t *c_pic,
                          uint32_t *d_cost)
{
    const int b_x=blockIdx.x;
    const int by=blockIdx.y;
    const int dis_n=threadIdx.x;

    const int bx=b_x%IMG_WIDTH;
    const int img_n=(b_x/IMG_WIDTH);
  //  const int img_p=(b_x/IMG_WIDTH)%4;

    uint32_t n_result=0;
    int dis_count,pixel_count,t_pixel_count;
    pixel_count=by*IMG_WIDTH+bx;
    dis_count=img_n*NEW_MAX_DISPARITY+dis_n;
    t_pixel_count=dis_count*(IMG_HEIGHT*IMG_WIDTH)+pixel_count;
    cost_t center_cost=center_transform[pixel_count];//get the center cos
      n_result=popcount(center_cost^l_c_transform[t_pixel_count]);

    //d_cost[(bx+by*IMG_WIDTH)*NEW_MAX_DISPARITY+dis_n]+=10*(n_result);
    d_cost[(bx+by*IMG_WIDTH)*NEW_MAX_DISPARITY+dis_n]+=(n_result);
}

__global__ void SumCost(uint32_t *l_cost,uint32_t *r_cost,uint32_t *t_cost,uint32_t *b_cost,uint32_t *sum_cost)
{
    int bx=blockIdx.x+1;
    int by=blockIdx.y+1;
    int tx=threadIdx.x;
    int t_num=blockDim.x;
    int index=(bx*by-1)*t_num+tx;
    sum_cost[index]=l_cost[index]+r_cost[index]+b_cost[index]+t_cost[index];
}
//****************negative disparity*******
__global__ void
N_HammingDistanceCostKernel1_Z(cost_t *l_transform1,
                               cost_t *r_transform1 ,
                               const cost_t* center_transform,
                               cost_t *t_transform1,
                               cost_t *b_transform1,
                               uint8_t *d_cost, const int rows, const int cols,const int image_number,const int sum_disparity)
{

    const int x=   blockIdx.x;  // the center image pixel x
    const int y=   blockIdx.y;  // the center image pixel y
    const int THRid = threadIdx.x; // represend the cost label

    //l_transfgorm b_transform is right top
    //cost_t center_tran=center_transform[y*cols+x];
    //float disparity=0.05*THRid*image_number;
    float disparity=0.05*THRid*image_number;
    int l_disparity=int(disparity);
    int r_disparity=(disparity-l_disparity>0)?(l_disparity+1):(l_disparity);
    float dif=disparity-l_disparity;
    int result=0;
    int n_result=0;
    int l_dif;
    int r_dif;
    cost_t center_cost=center_transform[x+y*cols];
    /*cost_t l_r_cost=(x+r_disparity)<cols?l_transform1[x+r_disparity+y*cols]:0;
    cost_t l_l_cost=(x+l_disparity)<cols?l_transform1[x+l_disparity+y*cols]:0;

    cost_t r_r_cost=(x-r_disparity)>=0?r_transform1[x-r_disparity+y*cols]:0;
    cost_t r_l_cost=(x-l_disparity)>=0?r_transform1[x-l_disparity+y*cols]:0;

    cost_t b_r_cost=(y+r_disparity)<rows?b_transform1[x+(y+r_disparity)*cols]:0;
    cost_t b_l_cost=(y+l_disparity)<rows?b_transform1[x+(y+l_disparity)*cols]:0;

    cost_t t_r_cost=(y-r_disparity)>=0?t_transform1[x+(y-r_disparity)*cols]:0;
    cost_t t_l_cost=(y-l_disparity)>=0?t_transform1[x+(y-l_disparity)*cols]:0;*/

    cost_t l_r_cost=(x+r_disparity)<cols?l_transform1[x+r_disparity+y*cols]:l_transform1[cols-1+y*cols];
    cost_t l_l_cost=(x+l_disparity)<cols?l_transform1[x+l_disparity+y*cols]:l_transform1[cols-1+y*cols];

    cost_t r_r_cost=(x-r_disparity)>=0?r_transform1[x-r_disparity+y*cols]:r_transform1[0+y*cols];
    cost_t r_l_cost=(x-l_disparity)>=0?r_transform1[x-l_disparity+y*cols]:r_transform1[0+y*cols];

    cost_t b_r_cost=(y+r_disparity)<rows?b_transform1[x+(y+r_disparity)*cols]:b_transform1[x+(rows-1)*cols];
    cost_t b_l_cost=(y+l_disparity)<rows?b_transform1[x+(y+l_disparity)*cols]:b_transform1[x+(rows-1)*cols];

    cost_t t_r_cost=(y-r_disparity)>=0?t_transform1[x+(y-r_disparity)*cols]:t_transform1[x+0*cols];
    cost_t t_l_cost=(y-l_disparity)>=0?t_transform1[x+(y-l_disparity)*cols]:t_transform1[x+0*cols];

    l_dif=int((1.0f-dif)*20.0f+0.1f);
    r_dif=int(dif*20.0f+0.1f);
    result+=(l_dif*((popcount(center_cost^l_l_cost)+popcount(center_cost^r_l_cost)+popcount(center_cost^b_l_cost)+popcount(center_cost^t_l_cost)))
             +r_dif*((popcount(center_cost^l_r_cost)+popcount(center_cost^r_r_cost)+popcount(center_cost^b_r_cost)+popcount(center_cost^t_r_cost))));

    //result/=10;
    //printf("%d\n",result);
   // d_cost[(x+y*cols)*NEW_MAX_DISPARITY+NEW_MAX_DISPARITY/2-1-THRid]=(uint8_t)result;
    result/=8;
    //if(result>255)
       // printf("%d::%d\n",result,result);
    d_cost[(x+y*cols)*NEW_MAX_DISPARITY+NEW_MAX_DISPARITY/2+THRid]=(uint8_t)(result);
    //***********positive d_cost finish******************************
    //***********negative d_cost begin*******************************

    /*l_r_cost=(x-r_disparity)>=0?l_transform1[x-r_disparity+y*cols]:0;
    l_l_cost=(x-l_disparity)>=0?l_transform1[x-l_disparity+y*cols]:0;

    r_r_cost=(x+r_disparity)<cols?r_transform1[x+r_disparity+y*cols]:0;
    r_l_cost=(x+l_disparity)<cols?r_transform1[x+l_disparity+y*cols]:0;

    b_r_cost=(y-r_disparity)>=0?b_transform1[x+(y-r_disparity)*cols]:0;
    b_l_cost=(y-l_disparity)>=0?b_transform1[x+(y-l_disparity)*cols]:0;

    t_r_cost=(y+r_disparity)<rows?t_transform1[x+(y+r_disparity)*cols]:0;
    t_l_cost=(y+l_disparity)<rows?t_transform1[x+(y+l_disparity)*cols]:0;*/
    l_r_cost=(x-r_disparity)>=0?l_transform1[x-r_disparity+y*cols]:l_transform1[0+y*cols];
    l_l_cost=(x-l_disparity)>=0?l_transform1[x-l_disparity+y*cols]:l_transform1[0+y*cols];

    r_r_cost=(x+r_disparity)<cols?r_transform1[x+r_disparity+y*cols]:r_transform1[cols-1+y*cols];
    r_l_cost=(x+l_disparity)<cols?r_transform1[x+l_disparity+y*cols]:r_transform1[cols-1+y*cols];

    b_r_cost=(y-r_disparity)>=0?b_transform1[x+(y-r_disparity)*cols]:b_transform1[x+0*cols];
    b_l_cost=(y-l_disparity)>=0?b_transform1[x+(y-l_disparity)*cols]:b_transform1[x+0*cols];

    t_r_cost=(y+r_disparity)<rows?t_transform1[x+(y+r_disparity)*cols]:t_transform1[x+(rows-1)*cols];
    t_l_cost=(y+l_disparity)<rows?t_transform1[x+(y+l_disparity)*cols]:t_transform1[x+(rows-1)*cols];

    float f1=(1.0-dif)*100*((popcount(center_cost^l_l_cost)+popcount(center_cost^r_l_cost)+popcount(center_cost^b_l_cost)+popcount(center_cost^t_l_cost)));
    float f2=dif*100*((popcount(center_cost^l_r_cost)+popcount(center_cost^r_r_cost)+popcount(center_cost^b_r_cost)+popcount(center_cost^t_r_cost)));


    n_result+=(l_dif*((popcount(center_cost^l_l_cost)+popcount(center_cost^r_l_cost)+popcount(center_cost^b_l_cost)+popcount(center_cost^t_l_cost)))
             +r_dif*((popcount(center_cost^l_r_cost)+popcount(center_cost^r_r_cost)+popcount(center_cost^b_r_cost)+popcount(center_cost^t_r_cost))));

    //n_result+=(f1+f2);
    //n_result/=10;
    //d_cost[(x+y*cols)*NEW_MAX_DISPARITY+NEW_MAX_DISPARITY/2+THRid]=(uint8_t)(n_result);
    //printf("%d::%d::%d\n",int((1.0f-dif)*20.0f+0.1f),int(dif*20.0f+0.1f),dif);
    n_result/=8;
   // if(n_result>255)
     //printf("%d\n",n_result);
    d_cost[(x+y*cols)*NEW_MAX_DISPARITY+NEW_MAX_DISPARITY/2-1-THRid]=(uint8_t)(n_result);

}
