#include "hip/hip_runtime.h"
/**
    This file is part of sgm. (https://github.com/dhernandez0/sgm).

    Copyright (c) 2016 Daniel Hernandez Juarez.

    sgm is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    sgm is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with sgm.  If not, see <http://www.gnu.org/licenses/>.

**/

#include "costs.h"
#include <stdio.h>
#include <>

__global__ void __launch_bounds__(1024, 2)
//CenterSymmetricCensusKernelSM2(const uint8_t *im, const uint8_t *im2, cost_t *transform, cost_t *transform2, const uint32_t rows, const uint32_t cols) {
CenterSymmetricCensusKernelSM2(const uint8_t *im,cost_t *transform,const uint32_t rows, const uint32_t cols) {
        const int idx = blockIdx.x*blockDim.x+threadIdx.x;
        const int idy = blockIdx.y*blockDim.y+threadIdx.y;

        const int win_cols = (32+LEFT*2); // 32+4*2 = 40
        const int win_rows = (32+TOP*2); // 32+3*2 = 38

        /*const int win_cols=7;
        const int win_rows=7;*/
        __shared__ uint8_t window[win_cols*win_rows];
        //__shared__ uint8_t window2[win_cols*win_rows];

        const int id = threadIdx.y*blockDim.x+threadIdx.x;
        const int sm_row = id / win_cols;
        const int sm_col = id % win_cols;

        const int im_row = blockIdx.y*blockDim.y+sm_row-TOP;
        const int im_col = blockIdx.x*blockDim.x+sm_col-LEFT;
        const bool boundaries = (im_row >= 0 && im_col >= 0 && im_row < rows && im_col < cols);
        //printf("fii\n");
        window[sm_row*win_cols+sm_col] = boundaries ? im[im_row*cols+im_col] : 0;
        //window2[sm_row*win_cols+sm_col] = boundaries ? im2[im_row*cols+im_col] : 0;

        // Not enough threads to fill window and window2
        const int block_size = blockDim.x*blockDim.y;
        if(id < (win_cols*win_rows-block_size)) {
            const int id = threadIdx.y*blockDim.x+threadIdx.x+block_size;
            const int sm_row = id / win_cols;
            const int sm_col = id % win_cols;

            const int im_row = blockIdx.y*blockDim.y+sm_row-TOP;
            const int im_col = blockIdx.x*blockDim.x+sm_col-LEFT;
            const bool boundaries = (im_row >= 0 && im_col >= 0 && im_row < rows && im_col < cols);
            window[sm_row*win_cols+sm_col] = boundaries ? im[im_row*cols+im_col] : 0;
            //window2[sm_row*win_cols+sm_col] = boundaries ? im2[im_row*cols+im_col] : 0;
        }

        __syncthreads();
        cost_t census = 0;
        //cost_t census2 = 0;
        if(idy < rows && idx < cols) {
            for(int k = 0; k < CENSUS_HEIGHT/2; k++) {
                for(int m = 0; m < CENSUS_WIDTH; m++) {
                    const uint8_t e1 = window[(threadIdx.y+k)*win_cols+threadIdx.x+m];
                    const uint8_t e2 = window[(threadIdx.y+2*TOP-k)*win_cols+threadIdx.x+2*LEFT-m];
                    //const uint8_t i1 = window2[(threadIdx.y+k)*win_cols+threadIdx.x+m];
                    //const uint8_t i2 = window2[(threadIdx.y+2*TOP-k)*win_cols+threadIdx.x+2*LEFT-m];

                    const int shft = k*CENSUS_WIDTH+m;
                    // Compare to the center
                    cost_t tmp = (e1 >= e2);
                    // Shift to the desired position
                    tmp <<= shft;
                    // Add it to its place
                    census |= tmp;
                    // Compare to the center
                    //cost_t tmp2 = (i1 >= i2);
                    // Shift to the desired position
                    //tmp2 <<= shft;
                    // Add it to its place
                    //census2 |= tmp2;
                }
            }
            if(CENSUS_HEIGHT % 2 != 0) {
                const int k = CENSUS_HEIGHT/2;
                for(int m = 0; m < CENSUS_WIDTH/2; m++) {
                    const uint8_t e1 = window[(threadIdx.y+k)*win_cols+threadIdx.x+m];
                    const uint8_t e2 = window[(threadIdx.y+2*TOP-k)*win_cols+threadIdx.x+2*LEFT-m];
                    //const uint8_t i1 = window2[(threadIdx.y+k)*win_cols+threadIdx.x+m];
                    //const uint8_t i2 = window2[(threadIdx.y+2*TOP-k)*win_cols+threadIdx.x+2*LEFT-m];

                    const int shft = k*CENSUS_WIDTH+m;
                    // Compare to the center
                    cost_t tmp = (e1 >= e2);
                    // Shift to the desired position
                    tmp <<= shft;
                    // Add it to its place
                    census |= tmp;
                    // Compare to the center
                    //cost_t tmp2 = (i1 >= i2);
                    // Shift to the desired position
                    //tmp2 <<= shft;
                    // Add it to its place
                    //census2 |= tmp2;
                }
            }

            transform[idy*cols+idx] = census;
            //transform2[idy*cols+idx] = census2;
        }
}


//__global__ void
//N_CenterSymmetricCnesusKernelSM2( uint8_t **l_im, uint8_t **r_im,cost_t **l_transform,cost_t **r_transform)
__global__ void N_CenterSymmetricCnesusKernelSM2( uint8_t **l_im,cost_t **l_transform)
{
    const int p_x=(IMG_WIDTH+blockDim.x-1) / blockDim.x;
    //printf("%d\n",p_x);
    const int bx=blockIdx.x/p_x;
    int img_num=bx;
    //if(img_num>=SELECT_IMAGE_NUM*NEW_MAX_DISPARITY)
    //if(img_num == 125)
    //printf("img_num::%d\n",img_num);
    C_CenterSymmetricCensusKernelSM2(l_im[img_num],l_transform[img_num],IMG_HEIGHT,IMG_WIDTH);
    //C_CenterSymmetricCensusKernelSM2(t_im[img_num],b_im[img_num],t_transform[img_num],b_transform[img_num],IMG_HEIGHT,IMG_WIDTH);
}

//__device__ void
 //C_CenterSymmetricCensusKernelSM2( uint8_t *im,  uint8_t *im2, cost_t *transform, cost_t *transform2, const uint32_t rows, const uint32_t cols)
__device__ void C_CenterSymmetricCensusKernelSM2( uint8_t *im, cost_t *transform, const uint32_t rows, const uint32_t cols)
 {
    //printf("begin\n");
     const int p_x=(IMG_WIDTH+blockDim.x-1) / blockDim.x;
    const int bx=blockIdx.x%(p_x);
    //if(bx>=16)
     //   printf("bx::%d\n",bx);
    const int idx = bx*blockDim.x+threadIdx.x;
    const int idy = blockIdx.y*blockDim.y+threadIdx.y;

    const int win_cols = (32+LEFT*2); // 32+4*2 = 40
    const int win_rows = (32+TOP*2); // 32+3*2 = 38

    /*const int win_cols=7;
    const int win_rows=7;*/
    __shared__ uint8_t window[win_cols*win_rows];
    //__shared__ uint8_t window2[win_cols*win_rows];

    const int id = threadIdx.y*blockDim.x+threadIdx.x;
    const int sm_row = id / win_cols;
    const int sm_col = id % win_cols;

    const int im_row = blockIdx.y*blockDim.y+sm_row-TOP;
    const int im_col = bx*blockDim.x+sm_col-LEFT;
    const bool boundaries = (im_row >= 0 && im_col >= 0 && im_row < rows && im_col < cols);
    //printf("fii\n");
    window[sm_row*win_cols+sm_col] = boundaries ? im[im_row*cols+im_col] : 0;
    //window2[sm_row*win_cols+sm_col] = boundaries ? im2[im_row*cols+im_col] : 0;

    // Not enough threads to fill window and window2
    const int block_size = blockDim.x*blockDim.y;
    if(id < (win_cols*win_rows-block_size)) {
        const int id = threadIdx.y*blockDim.x+threadIdx.x+block_size;
        const int sm_row = id / win_cols;
        const int sm_col = id % win_cols;

        const int im_row = blockIdx.y*blockDim.y+sm_row-TOP;
        const int im_col = bx*blockDim.x+sm_col-LEFT;
        const bool boundaries = (im_row >= 0 && im_col >= 0 && im_row < rows && im_col < cols);
        window[sm_row*win_cols+sm_col] = boundaries ? im[im_row*cols+im_col] : 0;
        //window2[sm_row*win_cols+sm_col] = boundaries ? im2[im_row*cols+im_col] : 0;
    }

    __syncthreads();
    cost_t census = 0;
    //cost_t census2 = 0;
    if(idy < rows && idx < cols) {
            for(int k = 0; k < CENSUS_HEIGHT/2; k++) {
                for(int m = 0; m < CENSUS_WIDTH; m++) {
                    const uint8_t e1 = window[(threadIdx.y+k)*win_cols+threadIdx.x+m];
                    const uint8_t e2 = window[(threadIdx.y+2*TOP-k)*win_cols+threadIdx.x+2*LEFT-m];
                    //const uint8_t i1 = window2[(threadIdx.y+k)*win_cols+threadIdx.x+m];
                    //const uint8_t i2 = window2[(threadIdx.y+2*TOP-k)*win_cols+threadIdx.x+2*LEFT-m];

                    const int shft = k*CENSUS_WIDTH+m;
                    // Compare to the center
                    cost_t tmp = (e1 >= e2);
                    // Shift to the desired position
                    tmp <<= shft;
                    // Add it to its place
                    census |= tmp;
                    // Compare to the center
                    //cost_t tmp2 = (i1 >= i2);
                    // Shift to the desired position
                    //tmp2 <<= shft;
                    // Add it to its place
                    //census2 |= tmp2;
                }
            }
            if(CENSUS_HEIGHT % 2 != 0) {
                const int k = CENSUS_HEIGHT/2;
                for(int m = 0; m < CENSUS_WIDTH/2; m++) {
                    const uint8_t e1 = window[(threadIdx.y+k)*win_cols+threadIdx.x+m];
                    const uint8_t e2 = window[(threadIdx.y+2*TOP-k)*win_cols+threadIdx.x+2*LEFT-m];
                    //const uint8_t i1 = window2[(threadIdx.y+k)*win_cols+threadIdx.x+m];
                    //const uint8_t i2 = window2[(threadIdx.y+2*TOP-k)*win_cols+threadIdx.x+2*LEFT-m];

                    const int shft = k*CENSUS_WIDTH+m;
                    // Compare to the center
                    cost_t tmp = (e1 >= e2);
                    // Shift to the desired position
                    tmp <<= shft;
                    // Add it to its place
                    census |= tmp;
                    // Compare to the center
                    //cost_t tmp2 = (i1 >= i2);
                    // Shift to the desired position
                    //tmp2 <<= shft;
                    // Add it to its place
                    //census2 |= tmp2;
                }
            }

        transform[idy*cols+idx] = census;
        //transform2[idy*cols+idx] = census2;
    }
}

__global__ void
ShiftImage(uint8_t **orign_im, uint8_t **center_warp,int px,int py)
{
    //printf("begin\n");
    //const int b_x=blockIdx.x;
    const int b_y=blockIdx.y;
    int image_n=b_y/NEW_MAX_DISPARITY;
    int dis_n=b_y-image_n*(NEW_MAX_DISPARITY);
    float d_dis=(dis_n+1-SHIFT_MIDDLE)*WARP_DIS;
    float d_y=py*d_dis*(image_n+1),d_x=px*d_dis*(image_n+1);
    WarpImage(orign_im[image_n],center_warp[b_y],d_x,d_y);
}
__global__ void N_ShiftImage(uint8_t **left_img,uint8_t **left_warp,uint8_t **right_img,uint8_t **right_warp,
                             uint8_t **top_img, uint8_t **top_warp,uint8_t **bottom_img,uint8_t **bottom_warp,
                             uint8_t **top_left_img, uint8_t **top_left_warp, uint8_t **top_right_img, uint8_t **top_right_warp,
                             uint8_t **bottom_left_img, uint8_t **bottom_left_warp, uint8_t **bottom_right_img, uint8_t **bottom_right_warp)
{
    const int b_y=blockIdx.y;
    /*if(b_y>64)
        printf("%d\n",b_y);*/
    const int t_y=threadIdx.y;
    int image_n=b_y/NEW_MAX_DISPARITY;
    int label_n = IMAGE_NUMBER-1-image_n;
    label_n =0;
    //int dis_n=b_y-image_n*(NEW_MAX_DISPARITY);
    int dis_n =  b_y-image_n*(NEW_MAX_DISPARITY);
    //float d_dis=(dis_n+1-SHIFT_MIDDLE)*WARP_DIS;
    //if(dis_n==127)
    //   printf("dis*****************%d\n",dis_n);
    const float warp_dis= ((float(END_DIS-BEGIN_DIS)/NEW_MAX_DISPARITY));
    float d_dis=(dis_n-SHIFT_MIDDLE)*warp_dis+BEGIN_DIS;
    //if(d_dis==WARP_DIS*-63)
    //  printf("d_dis::*****************%f\n",d_dis);
    // float d_y=py*d_dis*(image_n+1),d_x=px*d_dis*(image_n+1);
    int px,py;
    float d_x,d_y;
    //if(t_y==0)
    {
        px=-1,py=0;
        //d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        d_y=py*d_dis*(1),d_x=px*(d_dis*(1)+BEGIN_DIS);
        WarpImage(left_img[image_n],left_warp[b_y],d_x,d_y);
    }
    // else if(t_y==1)
    {
        px=1,py=0;
        //d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1)+BEGIN_DIS;
        d_y=py*d_dis*(label_n),d_x=px*(d_dis*(1)+BEGIN_DIS);
        /*if(d_x>64.0f)
            printf("%f::%f\n",d_y,d_x);*/
        WarpImage(right_img[image_n],right_warp[b_y],d_x,d_y);
    }
    //else if(t_y==2)
    {
        px=0,py=-1;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(top_img[image_n],top_warp[b_y],d_x,d_y);
    }
    //else
    {
        px=0,py=1;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(bottom_img[image_n],bottom_warp[b_y],d_x,d_y);
    }

    //top_left
    {
        px=-1, py=-1;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(top_left_img[image_n],top_left_warp[b_y],d_x,d_y);
    }
    //top_right
    {
        px=1, py=-1;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(top_right_img[image_n],top_right_warp[b_y],d_x,d_y);
    }
    //bottom_left
    {
        px=-1, py=1;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(bottom_left_img[image_n],bottom_left_warp[b_y],d_x,d_y);
    }
    //bottom_right
    {
        px=1, py=1;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(bottom_right_img[image_n],bottom_right_warp[b_y],d_x,d_y);
    }
}
/*__global__ void N_ShiftImage(uint8_t **left_img,uint8_t **left_warp,uint8_t **right_img,uint8_t **right_warp,
                             uint8_t **top_img, uint8_t **top_warp,uint8_t **bottom_img,uint8_t **bottom_warp,
                             uint8_t **top_left_img, uint8_t **top_left_warp, uint8_t **top_right_img, uint8_t **top_right_warp,
                             uint8_t **bottom_left_img, uint8_t **bottom_left_warp, uint8_t **bottom_right_img, uint8_t **bottom_right_warp)
{
    const int b_y=blockIdx.y;

    const int t_y=threadIdx.y;
    int image_n=b_y/NEW_MAX_DISPARITY;
    int label_n = IMAGE_NUMBER-1-image_n;
    label_n =0;
    //int dis_n=b_y-image_n*(NEW_MAX_DISPARITY);
    int dis_n =  b_y-image_n*(NEW_MAX_DISPARITY);
    //float d_dis=(dis_n+1-SHIFT_MIDDLE)*WARP_DIS;
    //if(dis_n==127)
     //   printf("dis*****************%d\n",dis_n);
    const float warp_dis= WARP_DIS*((float(END_DIS-BEGIN_DIS)/NEW_MAX_DISPARITY));
    float d_dis=(dis_n-SHIFT_MIDDLE)*warp_dis;
    //if(d_dis==WARP_DIS*-63)
      //  printf("d_dis::*****************%f\n",d_dis);
   // float d_y=py*d_dis*(image_n+1),d_x=px*d_dis*(image_n+1);
    int px,py;
    float d_x,d_y;
    //if(t_y==0)
    {
        px=-1,py=0;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(left_img[image_n],left_warp[b_y],d_x,d_y);
    }
   // else if(t_y==1)
    {
        px=1,py=0;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1)+BEGIN_DIS;

        WarpImage(right_img[image_n],right_warp[b_y],d_x,d_y);
    }
    //else if(t_y==2)
    {
        px=0,py=-1;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(top_img[image_n],top_warp[b_y],d_x,d_y);
    }
    //else
    {
        px=0,py=1;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(bottom_img[image_n],bottom_warp[b_y],d_x,d_y);
    }

    //top_left
    {
        px=-1, py=-1;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(top_left_img[image_n],top_left_warp[b_y],d_x,d_y);
    }
    //top_right
    {
        px=1, py=-1;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(top_right_img[image_n],top_right_warp[b_y],d_x,d_y);
    }
    //bottom_left
    {
        px=-1, py=1;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(bottom_left_img[image_n],bottom_left_warp[b_y],d_x,d_y);
    }
    //bottom_right
    {
        px=1, py=1;
        d_y=py*d_dis*(label_n+1),d_x=px*d_dis*(label_n+1);
        WarpImage(bottom_right_img[image_n],bottom_right_warp[b_y],d_x,d_y);
    }
}*/
__device__ void
WarpImage(const uint8_t *center_im,uint8_t *n_image,float dx,float dy)
{
  //  const int img_x=blockIdx.x*blockDim.x+threadIdx.x;
  //  const int img_y=blockIdx.y*blockDim.y+threadIdx.y;
    //printf("begin\n");
    int img_x=2*threadIdx.x;
    const int img_y=blockIdx.x;


    float n_x=img_x-dx,n_y=img_y-dy;
    //get the orign x and the orign y
    //printf("size::%d",sizeof(n_image)/sizeof(n_image[0]));
    if(n_x<FLOAT_EPS||n_x>IMG_WIDTH-1||n_y<FLOAT_EPS||n_y>IMG_HEIGHT-1)
        return ;
    int tl_x=(int)n_x,tl_y=(int)n_y,tr_x=n_x+1,tr_y=tl_y;
    int bl_x=tl_x,bl_y=tl_y+1,br_x=bl_x+1,br_y=bl_y;

    uint8_t tl_color=center_im[tl_x+tl_y*IMG_WIDTH];
    uint8_t tr_color=center_im[tr_x+tr_y*IMG_WIDTH];
    uint8_t bl_color=center_im[bl_x+bl_y*IMG_WIDTH];
    uint8_t br_color=center_im[br_x+br_y*IMG_WIDTH];

    uint8_t t_color=(tr_x-n_x)*tl_color+(n_x-tl_x)*tr_color;
    uint8_t b_color=(br_x-n_x)*bl_color+(n_x-bl_x)*br_color;
    int r_d=(br_y-n_y)*t_color+(n_y-tr_y)*b_color;
    uint8_t r_color=(br_y-n_y)*t_color+(n_y-tr_y)*b_color;
    n_image[img_x+img_y*IMG_WIDTH]=r_color;
    //n_image[0]=r_d;

    //fix for big size
    img_x = 2*threadIdx.x+1;
    n_x=img_x-dx,n_y=img_y-dy;
    //get the orign x and the orign y
    //printf("size::%d",sizeof(n_image)/sizeof(n_image[0]));
    if(n_x<FLOAT_EPS||n_x>IMG_WIDTH-1||n_y<FLOAT_EPS||n_y>IMG_HEIGHT-1)
        return ;
    tl_x=(int)n_x,tl_y=(int)n_y,tr_x=n_x+1,tr_y=tl_y;
    bl_x=tl_x,bl_y=tl_y+1,br_x=bl_x+1,br_y=bl_y;

    tl_color=center_im[tl_x+tl_y*IMG_WIDTH];
    tr_color=center_im[tr_x+tr_y*IMG_WIDTH];
    bl_color=center_im[bl_x+bl_y*IMG_WIDTH];
    br_color=center_im[br_x+br_y*IMG_WIDTH];

    t_color=(tr_x-n_x)*tl_color+(n_x-tl_x)*tr_color;
    b_color=(br_x-n_x)*bl_color+(n_x-bl_x)*br_color;
    r_d=(br_y-n_y)*t_color+(n_y-tr_y)*b_color;
    r_color=(br_y-n_y)*t_color+(n_y-tr_y)*b_color;
    n_image[img_x+img_y*IMG_WIDTH]=r_color;

}
